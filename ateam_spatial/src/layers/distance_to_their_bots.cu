#include "hip/hip_runtime.h"
#include "ateam_spatial/layers/width_of_shot_on_goal.hpp"
#include "ateam_spatial/layers/line_of_sight.hpp"
#include "ateam_spatial/coordinate_conversions.hpp"

namespace ateam_spatial::layers
{

CUDA_HOSTDEV float DistanceToTheirBots(const int x, const int y, const Robot * their_robots, const FieldDimensions & field_dims, const SpatialSettings & settings) {
  const auto real_x = SpatialToRealX(x, field_dims, settings);
  const auto real_y = SpatialToRealY(y, field_dims, settings);
  auto result = MAXFLOAT;
  for(auto i = 0; i < 16; ++i) {
    const auto & robot = their_robots[i];
    if(!robot.visible) {
      continue;
    }
    const auto distance = hypotf(robot.x - real_x, robot.y - real_y);
    result = min(result, distance);
  }
  return result;
}

} // namespace ateam_spatial::layers
