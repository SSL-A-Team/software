#include "hip/hip_runtime.h"
#include "ateam_spatial/layers/line_of_sight_ball.hpp"
#include "ateam_spatial/coordinate_conversions.hpp"

namespace ateam_spatial::layers
{

CUDA_HOSTDEV float PointDistance(const float x_1, const float y_1, const float x_2, const float y_2) {
  return hypotf((x_2 - x_1), (y_2 - y_1));
}

CUDA_HOSTDEV float DistanceToSegment(const float line_1_x, const float line_1_y, const float line_2_x, const float line_2_y, const float point_x, const float point_y)
{
  const auto l2 = PointDistance(line_1_x, line_1_y, line_2_x, line_2_y);
  if (fabsf(l2) < 1e-8f) {
    return PointDistance(line_1_x, line_1_y, point_x, point_y);
  }
  auto t = (((point_x - line_1_x) * (line_2_x - line_1_x)) + ((point_y - line_1_y)*(line_2_y - line_1_y))) / (l2*l2);
  t = max(0.0f, min(1.0f, t));
  const auto projected_x = line_1_x + (t * (line_2_x - line_1_x));
  const auto projected_y = line_1_y + (t * (line_2_y - line_1_y));
  return PointDistance(point_x, point_y, projected_x, projected_y);
}

CUDA_HOSTDEV float LineOfSight(const float src_x, const float src_y, const float dst_x, const float dst_y, const Robot * their_bots, const FieldDimensions & field_dims, const SpatialSettings & settings) {
  const auto robot_radius = 0.09f;

  bool result = 1;
  for(auto i = 0; i < 16; ++i) {
    const auto & robot = their_bots[i];
    if(!robot.visible) {
      continue;
    }
    const auto distance = DistanceToSegment(dst_x, dst_y, src_x, src_y, robot.x, robot.y);
    result &= distance > robot_radius;
  }

  return result;
}

} // namespace ateam_spatial::layers
