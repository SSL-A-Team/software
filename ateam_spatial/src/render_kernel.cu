#include "hip/hip_runtime.h"
// Copyright 2025 A Team
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.

#include "ateam_spatial/render_kernel.hpp"
#include <cstdio>
#include <cmath>

namespace ateam_spatial
{

__global__ void render_kernel(const float * input_buffer, const std::size_t buffer_size,
                              const float min_value, const float max_value,
                              uint8_t * output_buffer)
{
  const auto index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index >= buffer_size) {
    return;
  }

  output_buffer[index] = (input_buffer[index] - min_value) * (255 / (max_value - min_value));
}

}  // namespace ateam_spatial
